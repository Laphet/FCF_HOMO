#include "hip/hip_runtime.h"
#include "cuda-fct-solver.hpp"

#define MAX_THREADS_PER_BLOCK  1024
#define WARP_SIZE              32
#define FCT_POST_STENCIL_WIDTH 2
#define IFCT_PRE_STENCIL_WIDTH 4

void check(hipError_t status, char const *const func, char const *const file, int const line)
{
  if (status != hipSuccess) {
    std::cerr << "CUDA Runtime Error at: " << file << ":" << line << std::endl;
    std::cerr << hipGetErrorString(status) << " " << func << std::endl;
    std::exit(EXIT_FAILURE);
  }
}

void check(hipfftResult status, char const *const func, char const *const file, int const line)
{
  if (status != HIPFFT_SUCCESS) {
    std::cerr << "cufft Error at: " << file << ":" << line << std::endl;
    std::cerr << "with code (" << status << ") " << func << std::endl;
    std::exit(EXIT_FAILURE);
  }
}

void check(hipsparseStatus_t status, char const *const func, char const *const file, int const line)
{
  if (status != HIPSPARSE_STATUS_SUCCESS) {
    std::cerr << "cusparse Error at: " << file << ":" << line << std::endl;
    std::cerr << hipsparseGetErrorString(status) << " " << func << std::endl;
    std::exit(EXIT_FAILURE);
  }
}

void check(hipblasStatus_t status, char const *const func, char const *const file, int const line)
{
  if (status != HIPBLAS_STATUS_SUCCESS) {
    std::cerr << "cusparse Error at: " << file << ":" << line << std::endl;
    std::cerr << "with code (" << status << ") " << func << std::endl;
    std::exit(EXIT_FAILURE);
  }
}

void checkLast(char const *const file, int const line)
{
  hipError_t err{hipGetLastError()};
  if (err != hipSuccess) {
    std::cerr << "CUDA Runtime Error at: " << file << ":" << line << std::endl;
    std::cerr << hipGetErrorString(err) << std::endl;
    std::exit(EXIT_FAILURE);
  }
}

__device__ int getIdxFrom3dIdx_d(const int i, const int j, const int k, const int N, const int P)
{
  return i * N * P + (j * P) + k;
}

__device__ int getIdxFrom3dIdxHalf(const int i, const int j, const int k, const int N, const int P)
{
  return i * (N / 2 + 1) * P + (j * P) + k;
}

/*
    Note that P may not be a 32x integer, which my cause warp divergences.
    Hence this routine is designed to make every 32 threads operate the
    same i and j.
*/

__host__ __device__ int getPmod(const int P)
{
  return (P / WARP_SIZE + 1) * WARP_SIZE;
}

__device__ void get3dIdxFromThreadIdx(int &i, int &j, int &k, const int glbThreadIdx, const int N, const int P, const int Pmod)
{
  i = glbThreadIdx / (N * Pmod);
  j = (glbThreadIdx / Pmod) % N;
  k = (glbThreadIdx % Pmod) % P;
}

__device__ hipComplex getExpItheta(const float theta)
{
  return make_hipComplex(cosf(theta), sinf(theta));
}

__device__ hipDoubleComplex getExpItheta(const double theta)
{
  return make_hipDoubleComplex(cos(theta), sin(theta));
}

__device__ hipComplex hipCmul(hipComplex cVar1, hipComplex cVar2)
{
  return hipCmulf(cVar1, cVar2);
}

template <typename T>
__global__ void fctPre(T *out, const T *in, const int M, const int N, const int P)
{
  size_t       glbThreadIdx{blockIdx.x * blockDim.x + threadIdx.x};
  int          i{0}, j{0}, k{0};
  int          Pmod{getPmod(P)};
  __shared__ T in_buffer[MAX_THREADS_PER_BLOCK];

  if (glbThreadIdx < M * N * Pmod) {
    int idx_req{0};
    get3dIdxFromThreadIdx(i, j, k, glbThreadIdx, N, P, Pmod);

    if (i < (M + 1) / 2 && j < (N + 1) / 2) idx_req = getIdxFrom3dIdx_d(2 * i, 2 * j, k, N, P);
    if ((M + 1) / 2 <= i && j < (N + 1) / 2) idx_req = getIdxFrom3dIdx_d(2 * M - 2 * i - 1, 2 * j, k, N, P);
    if (i < (M + 1) / 2 && (N + 1) / 2 <= j) idx_req = getIdxFrom3dIdx_d(2 * i, 2 * N - 2 * j - 1, k, N, P);
    if ((M + 1) / 2 <= i && (N + 1) / 2 <= j) idx_req = getIdxFrom3dIdx_d(2 * M - 2 * i - 1, 2 * N - 2 * j - 1, k, N, P);

    in_buffer[threadIdx.x] = in[idx_req];
  }
  __syncthreads();

  if (glbThreadIdx < M * N * Pmod) {
    int idx_tar{getIdxFrom3dIdx_d(i, j, k, N, P)};
    out[idx_tar] = in_buffer[threadIdx.x];
  }
}

template <typename T>
__global__ void fctPost(T *out_hat, const decltype(cuTraits<T>::compVar) *in_hat, const int M, const int N, const int P)
{
  using complex_T = decltype(cuTraits<T>::compVar);
  size_t       glbThreadIdx{blockIdx.x * blockDim.x + threadIdx.x};
  int          i_p{0}, j_p{0}, k{0};
  int          Pmod{getPmod(P)};
  __shared__ T in_hat_buffer[2 * FCT_POST_STENCIL_WIDTH][MAX_THREADS_PER_BLOCK + 1];
  // Cannot use cuda::std::complex<T> here.
  // Avoid bank conflicts, we add a padding to every row here.

  if (glbThreadIdx < M * N * Pmod) {
    get3dIdxFromThreadIdx(i_p, j_p, k, glbThreadIdx, N, P, Pmod);
    int idx_req{0};
    if (1 <= i_p && j_p < N / 2 + 1) {
      idx_req                       = getIdxFrom3dIdxHalf(i_p, j_p, k, N, P);
      in_hat_buffer[0][threadIdx.x] = in_hat[idx_req].x;
      in_hat_buffer[1][threadIdx.x] = in_hat[idx_req].y;

      idx_req                       = getIdxFrom3dIdxHalf(M - i_p, j_p, k, N, P);
      in_hat_buffer[2][threadIdx.x] = in_hat[idx_req].x;
      in_hat_buffer[3][threadIdx.x] = in_hat[idx_req].y;
    }
    if (0 == i_p && j_p < N / 2 + 1) {
      idx_req                       = getIdxFrom3dIdxHalf(0, j_p, k, N, P);
      in_hat_buffer[0][threadIdx.x] = in_hat[idx_req].x;
      in_hat_buffer[1][threadIdx.x] = in_hat[idx_req].y;

      idx_req                       = getIdxFrom3dIdxHalf(0, j_p, k, N, P);
      in_hat_buffer[2][threadIdx.x] = in_hat[idx_req].x;
      in_hat_buffer[3][threadIdx.x] = in_hat[idx_req].y;
    }
    if (1 <= i_p && N / 2 + 1 <= j_p) {
      idx_req                       = getIdxFrom3dIdxHalf(M - i_p, N - j_p, k, N, P);
      in_hat_buffer[0][threadIdx.x] = in_hat[idx_req].x;
      in_hat_buffer[1][threadIdx.x] = in_hat[idx_req].y;

      idx_req                       = getIdxFrom3dIdxHalf(i_p, N - j_p, k, N, P);
      in_hat_buffer[2][threadIdx.x] = in_hat[idx_req].x;
      in_hat_buffer[3][threadIdx.x] = in_hat[idx_req].y;
    }
    if (0 == i_p && N / 2 + 1 <= j_p) {
      idx_req                       = getIdxFrom3dIdxHalf(0, N - j_p, k, N, P);
      in_hat_buffer[0][threadIdx.x] = in_hat[idx_req].x;
      in_hat_buffer[1][threadIdx.x] = in_hat[idx_req].y;

      idx_req                       = getIdxFrom3dIdxHalf(0, N - j_p, k, N, P);
      in_hat_buffer[2][threadIdx.x] = in_hat[idx_req].x;
      in_hat_buffer[3][threadIdx.x] = in_hat[idx_req].y;
    }
  }
  __syncthreads();

  if (glbThreadIdx < M * N * Pmod) {
    int       idx_tar{getIdxFrom3dIdx_d(i_p, j_p, k, N, P)};
    T         i_theta, j_theta, cuPi{static_cast<T>(M_PI)}, temp0, temp1;
    complex_T ninj_exp, nipj_exp, tempBuff0, tempBuff1;
    i_theta  = static_cast<T>(i_p) / static_cast<T>(2 * M) * cuPi;
    j_theta  = static_cast<T>(j_p) / static_cast<T>(2 * N) * cuPi;
    ninj_exp = getExpItheta(-i_theta - j_theta);
    nipj_exp = getExpItheta(-i_theta + j_theta);

    if (1 <= j_p && j_p < N / 2 + 1) {
      tempBuff0.x      = in_hat_buffer[0][threadIdx.x];
      tempBuff0.y      = in_hat_buffer[1][threadIdx.x];
      temp0            = ninj_exp.x * tempBuff0.x - ninj_exp.y * tempBuff0.y;
      tempBuff1.x      = in_hat_buffer[2][threadIdx.x];
      tempBuff1.y      = -in_hat_buffer[3][threadIdx.x];
      temp1            = nipj_exp.x * tempBuff1.x - nipj_exp.y * tempBuff1.y;
      out_hat[idx_tar] = (temp0 + temp1) / 2;
      return;
    }
    if (N / 2 + 1 <= j_p) {
      tempBuff0.x      = in_hat_buffer[0][threadIdx.x];
      tempBuff0.y      = -in_hat_buffer[1][threadIdx.x];
      temp0            = ninj_exp.x * tempBuff0.x - ninj_exp.y * tempBuff0.y;
      tempBuff1.x      = in_hat_buffer[2][threadIdx.x];
      tempBuff1.y      = in_hat_buffer[3][threadIdx.x];
      temp1            = nipj_exp.x * tempBuff1.x - nipj_exp.y * tempBuff1.y;
      out_hat[idx_tar] = (temp0 + temp1) / 2;
      return;
    }
    if (0 == j_p) {
      tempBuff0.x      = in_hat_buffer[0][threadIdx.x];
      tempBuff0.y      = in_hat_buffer[1][threadIdx.x];
      out_hat[idx_tar] = ninj_exp.x * tempBuff0.x - ninj_exp.y * tempBuff0.y;
      return;
    }
  } else return;
}

template <typename T>
__global__ void ifctPre(decltype(cuTraits<T>::compVar) *out_hat, const T *in_hat, const int M, const int N, const int P)
{
  using complex_T = decltype(cuTraits<T>::compVar);
  size_t       glbThreadIdx{blockIdx.x * blockDim.x + threadIdx.x};
  int          i_p{0}, j_p{0}, k{0};
  int          Pmod{(P / WARP_SIZE + 1) * WARP_SIZE};
  __shared__ T in_hat_buffer[IFCT_PRE_STENCIL_WIDTH][MAX_THREADS_PER_BLOCK + 1];
  /* Avoid bank conflicts, we add a pad to every row here. */

  if (glbThreadIdx < M * N * Pmod) {
    get3dIdxFromThreadIdx(i_p, j_p, k, glbThreadIdx, N, P, Pmod);
    int idx_req{getIdxFrom3dIdx_d(i_p, j_p, k, N, P)};
    in_hat_buffer[0][threadIdx.x] = in_hat[idx_req];
    if (0 < i_p && 0 < j_p) {
      idx_req                       = getIdxFrom3dIdx_d(M - i_p, N - j_p, k, N, P);
      in_hat_buffer[1][threadIdx.x] = in_hat[idx_req];

      idx_req                       = getIdxFrom3dIdx_d(M - i_p, j_p, k, N, P);
      in_hat_buffer[2][threadIdx.x] = in_hat[idx_req];

      idx_req                       = getIdxFrom3dIdx_d(i_p, N - j_p, k, N, P);
      in_hat_buffer[3][threadIdx.x] = in_hat[idx_req];
    }
    if (0 == i_p && 0 < j_p) {
      in_hat_buffer[1][threadIdx.x] = 0;

      in_hat_buffer[2][threadIdx.x] = 0;

      idx_req                       = getIdxFrom3dIdx_d(0, N - j_p, k, N, P);
      in_hat_buffer[3][threadIdx.x] = in_hat[idx_req];
    }
    if (0 < i_p && 0 == j_p) {
      in_hat_buffer[1][threadIdx.x] = 0;

      idx_req                       = getIdxFrom3dIdx_d(M - i_p, 0, k, N, P);
      in_hat_buffer[2][threadIdx.x] = in_hat[idx_req];

      in_hat_buffer[3][threadIdx.x] = 0;
    }
    if (0 == i_p && 0 == j_p) {
      in_hat_buffer[1][threadIdx.x] = 0;

      in_hat_buffer[2][threadIdx.x] = 0;

      in_hat_buffer[3][threadIdx.x] = 0;
    }
  }
  __syncthreads();

  if (glbThreadIdx < M * N * Pmod && j_p < N / 2 + 1) {
    T         i_theta, j_theta, cuPi{static_cast<T>(M_PI)};
    complex_T temp, pipj_exp;
    i_theta  = static_cast<T>(i_p) / static_cast<T>(2 * M) * cuPi;
    j_theta  = static_cast<T>(j_p) / static_cast<T>(2 * N) * cuPi;
    pipj_exp = getExpItheta(i_theta + j_theta);
    temp.x   = in_hat_buffer[0][threadIdx.x] - in_hat_buffer[1][threadIdx.x];
    temp.y   = -(in_hat_buffer[2][threadIdx.x] + in_hat_buffer[3][threadIdx.x]);
    int idx_tar{getIdxFrom3dIdxHalf(i_p, j_p, k, N, P)};
    out_hat[idx_tar] = hipCmul(pipj_exp, temp);
    return;
  } else return;
}

template <typename T>
__global__ void ifctPost(T *out, const T *in, const int M, const int N, const int P)
{
  size_t       glbThreadIdx{blockIdx.x * blockDim.x + threadIdx.x};
  int          i{0}, j{0}, k{0};
  int          Pmod{(P / WARP_SIZE + 1) * WARP_SIZE};
  __shared__ T in_buffer[MAX_THREADS_PER_BLOCK];

  if (glbThreadIdx < M * N * Pmod) {
    get3dIdxFromThreadIdx(i, j, k, glbThreadIdx, N, P, Pmod);
    int idx_req{0};
    if (0 == i % 2 && 0 == j % 2) idx_req = getIdxFrom3dIdx_d(i / 2, j / 2, k, N, P);
    if (0 == i % 2 && 1 == j % 2) idx_req = getIdxFrom3dIdx_d(i / 2, N - (j + 1) / 2, k, N, P);
    if (1 == i % 2 && 0 == j % 2) idx_req = getIdxFrom3dIdx_d(M - (i + 1) / 2, j / 2, k, N, P);
    if (1 == i % 2 && 1 == j % 2) idx_req = getIdxFrom3dIdx_d(M - (i + 1) / 2, N - (j + 1) / 2, k, N, P);

    in_buffer[threadIdx.x] = in[idx_req];
  }
  __syncthreads();

  if (glbThreadIdx < M * N * Pmod) {
    int     idx_tar{getIdxFrom3dIdx_d(i, j, k, N, P)};
    const T scalFactor{static_cast<T>(1) / (M * N)};
    /* cuFFT performs un-normalized FFTs! */
    out[idx_tar] = in_buffer[threadIdx.x] * scalFactor;
  }
}

template <typename T>
cufctSolver<T>::cufctSolver(const int _M, const int _N, const int _P) : dims{_M, _N, _P}, dlPtr{nullptr}, dPtr{nullptr}, duPtr{nullptr}, tridSolverBuffer{nullptr}, csrMat{nullptr, nullptr, nullptr, nullptr}
{
  CHECK_CUDA_ERROR(hipMalloc(reinterpret_cast<void **>(&realBuffer), sizeof(T) * _M * _N * _P));
  CHECK_CUDA_ERROR(hipMalloc(reinterpret_cast<void **>(&compBuffer), sizeof(cuCompType) * _M * (_N / 2 + 1) * _P));
  /* Thanks to the C2R transformation, we could use less memory for complex vectors. */
  /* Works on the cufft context. */
  int dimsHalf[3]{_M, _N / 2 + 1, _P};

  CHECK_CUDA_ERROR(hipfftCreate(&r2cPlan));
  CHECK_CUDA_ERROR(hipfftPlanMany(&r2cPlan, 2, &dims[0], &dims[0], _P, 1, &dimsHalf[0], _P, 1, cuTraits<T>::r2cType, _P));
  /* It is strange that cufft does not explain why need dimsHalf here. */
  /* There is some mismatch between fftw and cufft. */
  CHECK_CUDA_ERROR(hipfftCreate(&c2rPlan));
  CHECK_CUDA_ERROR(hipfftPlanMany(&c2rPlan, 2, &dims[0], &dimsHalf[0], _P, 1, &dims[0], _P, 1, cuTraits<T>::c2rType, _P));
  // CHECK_CUDA_ERROR(hipfftCreate(&c2cPlan));
  // CHECK_CUDA_ERROR(hipfftPlanMany(&c2rPlan, 2, &dims[0], &dims[0], dims[2], 1, &dims[0], dims[2], 1, cuTraits<T>::c2cType, dims[2]));

  /* Works on the cusparse cublas context. */
  CHECK_CUDA_ERROR(hipsparseCreate(&sprHandle));
  CHECK_CUDA_ERROR(hipblasCreate(&blasHandle));
}

hipfftResult cufftReal2Comp(hipfftHandle plan, float *idata, hipComplex *odata)
{
  return hipfftExecR2C(plan, reinterpret_cast<hipfftReal *>(idata), reinterpret_cast<hipfftComplex *>(odata));
}

hipfftResult cufftReal2Comp(hipfftHandle plan, double *idata, hipDoubleComplex *odata)
{
  return hipfftExecD2Z(plan, reinterpret_cast<hipfftDoubleReal *>(idata), reinterpret_cast<hipfftDoubleComplex *>(odata));
}

template <typename T>
void viewRealVec(std::vector<T> &vec)
{
  for (int i{0}; i < vec.size(); ++i) std::cout << '[' << i << "]=" << vec[i] << ' ';
  std::cout << '\n';
}

template <typename T>
void viewCompVec(std::vector<decltype(cuTraits<T>::compVar)> &vec)
{
  for (int i{0}; i < vec.size(); ++i) {
    std::cout << '[' << i << "]=" << vec[i].x;
    if (vec[i].y >= 0) std::cout << '+';
    std::cout << vec[i].y << "I\n";
  }
  std::cout << '\n';
}

template <typename T>
void cufctSolver<T>::fctForward(T *v)
{
  int M{dims[0]}, N{dims[1]}, P{dims[2]};
  int blockSize{0};   // The launch configurator returned block size
  int minGridSize{0}; // The minimum grid size needed to achieve the maximum occupancy for a full device launch
  int gridSize{0};    // The actual grid size needed, based on input size
  int Pmod{getPmod(P)};

  // std::vector<T>                              reViewer(M * N * P);
  // std::vector<decltype(cuTraits<T>::compVar)> cmViewer(M * (N / 2 + 1) * P);
  // std::cout << std::fixed << std::showpoint << std::setprecision(3);

  // CHECK_CUDA_ERROR(hipMemcpy(&reViewer[0], &v[0], M * N * P * sizeof(T), hipMemcpyDeviceToHost));
  // viewRealVec(reViewer);

  CHECK_CUDA_ERROR(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, &fctPre<T>, 0, 0));
  blockSize = (blockSize / WARP_SIZE) * WARP_SIZE; // This should be useless.
  if (blockSize < P) {
    std::cout << "Recommended blocksize=" << blockSize << " < P=" << P << ", reset blocksize=" << MAX_THREADS_PER_BLOCK << '\n';
    blockSize = MAX_THREADS_PER_BLOCK;
  }
  gridSize = (M * N * Pmod + blockSize - 1) / blockSize;
  fctPre<T><<<gridSize, blockSize>>>(&realBuffer[0], &v[0], M, N, P);
  CHECK_LAST_CUDA_ERROR();

  // CHECK_CUDA_ERROR(hipMemcpy(&reViewer[0], &realBuffer[0], M * N * P * sizeof(T), hipMemcpyDeviceToHost));
  // viewRealVec(reViewer);

  CHECK_CUDA_ERROR(cufftReal2Comp(r2cPlan, &realBuffer[0], &compBuffer[0]));

  // CHECK_CUDA_ERROR(hipMemcpy(&cmViewer[0], &compBuffer[0], M * (N / 2 + 1) * P * sizeof(cuCompType), hipMemcpyDeviceToHost));
  // viewCompVec<T>(cmViewer);

  CHECK_CUDA_ERROR(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, &fctPost<T>, 0, 0));
  blockSize = (blockSize / WARP_SIZE) * WARP_SIZE; // This should be useless.
  if (blockSize < P) {
    std::cout << "Recommended blocksize=" << blockSize << " < P=" << P << ", reset blocksize=" << MAX_THREADS_PER_BLOCK << '\n';
    blockSize = MAX_THREADS_PER_BLOCK;
  }
  gridSize = (M * N * Pmod + blockSize - 1) / blockSize;
  fctPost<T><<<gridSize, blockSize>>>(&v[0], &compBuffer[0], M, N, P);
  CHECK_LAST_CUDA_ERROR();
}

hipfftResult cufftComp2Real(hipfftHandle plan, hipComplex *idata, float *odata)
{
  return hipfftExecC2R(plan, reinterpret_cast<hipfftComplex *>(idata), reinterpret_cast<hipfftReal *>(odata));
}

hipfftResult cufftComp2Real(hipfftHandle plan, hipDoubleComplex *idata, double *odata)
{
  return hipfftExecZ2D(plan, reinterpret_cast<hipfftDoubleComplex *>(idata), reinterpret_cast<hipfftDoubleReal *>(odata));
}

template <typename T>
void cufctSolver<T>::fctBackward(T *v)
{
  int M{dims[0]}, N{dims[1]}, P{dims[2]};
  int blockSize{0};   // The launch configurator returned block size
  int minGridSize{0}; // The minimum grid size needed to achieve the maximum occupancy for a full device launch
  int gridSize{0};    // The actual grid size needed, based on input size
  int Pmod{(P / WARP_SIZE + 1) * WARP_SIZE};

  // std::vector<T>                              reViewer(M * N * P);
  // std::vector<decltype(cuTraits<T>::compVar)> cmViewer(M * (N / 2 + 1) * P);
  // std::cout << std::fixed << std::showpoint << std::setprecision(3);

  // CHECK_CUDA_ERROR(hipMemcpy(&reViewer[0], &v[0], M * N * P * sizeof(T), hipMemcpyDeviceToHost));
  // viewRealVec(reViewer);

  CHECK_CUDA_ERROR(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, &ifctPre<T>, 0, 0));
  blockSize = (blockSize / WARP_SIZE) * WARP_SIZE; // This should be useless.
  if (blockSize < P) {
    std::cout << "Recommended blocksize=" << blockSize << " < P=" << P << ", reset blocksize=" << MAX_THREADS_PER_BLOCK << '\n';
    blockSize = MAX_THREADS_PER_BLOCK;
  }
  gridSize = (M * N * Pmod + blockSize - 1) / blockSize;
  ifctPre<T><<<gridSize, blockSize>>>(&compBuffer[0], &v[0], M, N, P);
  CHECK_LAST_CUDA_ERROR();

  // CHECK_CUDA_ERROR(hipMemcpy(&cmViewer[0], &compBuffer[0], M * (N / 2 + 1) * P * sizeof(cuCompType), hipMemcpyDeviceToHost));
  // viewCompVec<T>(cmViewer);

  CHECK_CUDA_ERROR(cufftComp2Real(c2rPlan, &compBuffer[0], &realBuffer[0]));

  // CHECK_CUDA_ERROR(hipMemcpy(&reViewer[0], &realBuffer[0], M * N * P * sizeof(T), hipMemcpyDeviceToHost));
  // viewRealVec(reViewer);

  CHECK_CUDA_ERROR(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, &ifctPost<T>, 0, 0));
  blockSize = (blockSize / WARP_SIZE) * WARP_SIZE; // This should be useless.
  if (blockSize < P) {
    std::cout << "Recommended blocksize=" << blockSize << " < P=" << P << ", reset blocksize=" << MAX_THREADS_PER_BLOCK << '\n';
    blockSize = MAX_THREADS_PER_BLOCK;
  }
  gridSize = (M * N * Pmod + blockSize - 1) / blockSize;
  ifctPost<T><<<gridSize, blockSize>>>(&v[0], &realBuffer[0], M, N, P);
  CHECK_LAST_CUDA_ERROR();

  // CHECK_CUDA_ERROR(hipMemcpy(&reViewer[0], &v[0], M * N * P * sizeof(T), hipMemcpyDeviceToHost));
  // viewRealVec(reViewer);
}

void gtsv2StridedBatch_bufferSizeExt(hipsparseHandle_t handle, int m, const float *dl, const float *d, const float *du, const float *x, int batchCount, int batchStride, size_t *bufferSizeInBytes)
{
  CHECK_CUDA_ERROR(hipsparseSgtsv2StridedBatch_bufferSizeExt(handle, m, dl, d, du, x, batchCount, batchStride, bufferSizeInBytes));
}

void gtsv2StridedBatch_bufferSizeExt(hipsparseHandle_t handle, int m, const double *dl, const double *d, const double *du, const double *x, int batchCount, int batchStride, size_t *bufferSizeInBytes)
{
  CHECK_CUDA_ERROR(hipsparseDgtsv2StridedBatch_bufferSizeExt(handle, m, dl, d, du, x, batchCount, batchStride, bufferSizeInBytes));
}

template <typename T>
void cufctSolver<T>::setTridSolverData(T *dl, T *d, T *du)
{
  size_t size = dims[0] * dims[1] * dims[2];

  if (dlPtr != nullptr || dPtr != nullptr || duPtr != nullptr) std::cerr << "The internal data have been initialized, be careful!\n";

  if (dlPtr == nullptr) CHECK_CUDA_ERROR(hipMalloc(reinterpret_cast<void **>(&dlPtr), size * sizeof(T)));
  if (dPtr == nullptr) CHECK_CUDA_ERROR(hipMalloc(reinterpret_cast<void **>(&dPtr), size * sizeof(T)));
  if (duPtr == nullptr) CHECK_CUDA_ERROR(hipMalloc(reinterpret_cast<void **>(&duPtr), size * sizeof(T)));

  CHECK_CUDA_ERROR(hipMemcpy(reinterpret_cast<void *>(dlPtr), reinterpret_cast<void *>(dl), size * sizeof(T), hipMemcpyHostToDevice));
  CHECK_CUDA_ERROR(hipMemcpy(reinterpret_cast<void *>(dPtr), reinterpret_cast<void *>(d), size * sizeof(T), hipMemcpyHostToDevice));
  CHECK_CUDA_ERROR(hipMemcpy(reinterpret_cast<void *>(duPtr), reinterpret_cast<void *>(du), size * sizeof(T), hipMemcpyHostToDevice));

  size_t bufferSizeInBytes{0};
  int    M{dims[0]}, N{dims[1]}, P{dims[2]};
  gtsv2StridedBatch_bufferSizeExt(sprHandle, P, dlPtr, dPtr, duPtr, realBuffer, M * N, P, &bufferSizeInBytes);
  if (tridSolverBuffer != nullptr) std::cerr << "The internal data have been initialized, be careful!\n";
  CHECK_CUDA_ERROR(hipMalloc(reinterpret_cast<void **>(&tridSolverBuffer), bufferSizeInBytes));
}

void gtsv2StridedBatch(hipsparseHandle_t handle, int m, const float *dl, const float *d, const float *du, float *x, int batchCount, int batchStride, void *pBuffer)
{
  CHECK_CUDA_ERROR(hipsparseSgtsv2StridedBatch(handle, m, dl, d, du, x, batchCount, batchStride, pBuffer));
}

void gtsv2StridedBatch(hipsparseHandle_t handle, int m, const double *dl, const double *d, const double *du, double *x, int batchCount, int batchStride, void *pBuffer)
{
  CHECK_CUDA_ERROR(hipsparseDgtsv2StridedBatch(handle, m, dl, d, du, x, batchCount, batchStride, pBuffer));
}

template <typename T>
void cufctSolver<T>::precondSolver(T *rhs)
{
  if (dlPtr == nullptr || dPtr == nullptr || duPtr == nullptr || tridSolverBuffer == nullptr) {
    std::cerr << "The internal data have not been initialized!\n";
    std::cerr << "There will be nothing to do in this routine.\n";
    return;
  }

  fctForward(rhs);

  int M{dims[0]}, N{dims[1]}, P{dims[2]};
  gtsv2StridedBatch(sprHandle, P, dlPtr, dPtr, duPtr, rhs, M * N, P, tridSolverBuffer);

  fctBackward(rhs);
}

template <typename T>
void cufctSolver<T>::setSprMatData(int *csrRowOffsets, int *csrColInd, T *csrValues)
{
  int size{dims[0] * dims[1] * dims[2]};
  int nnz{csrRowOffsets[size]};

  if (csrMat.rowOffsetsPtr != nullptr || csrMat.colIndPtr != nullptr || csrMat.valuesPtr != nullptr) std::cerr << "The internal data have been initialized, be careful!\n";

  if (csrMat.rowOffsetsPtr == nullptr) CHECK_CUDA_ERROR(hipMalloc(reinterpret_cast<void **>(&csrMat.rowOffsetsPtr), (size + 1) * sizeof(int)));
  if (csrMat.colIndPtr == nullptr) CHECK_CUDA_ERROR(hipMalloc(reinterpret_cast<void **>(&csrMat.colIndPtr), nnz * sizeof(int)));
  if (csrMat.valuesPtr == nullptr) CHECK_CUDA_ERROR(hipMalloc(reinterpret_cast<void **>(&csrMat.valuesPtr), nnz * sizeof(T)));

  CHECK_CUDA_ERROR(hipMemcpy(reinterpret_cast<void *>(csrMat.rowOffsetsPtr), reinterpret_cast<void *>(csrRowOffsets), (size + 1) * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA_ERROR(hipMemcpy(reinterpret_cast<void *>(csrMat.colIndPtr), reinterpret_cast<void *>(csrColInd), nnz * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA_ERROR(hipMemcpy(reinterpret_cast<void *>(csrMat.valuesPtr), reinterpret_cast<void *>(csrValues), nnz * sizeof(T), hipMemcpyHostToDevice));

  CHECK_CUDA_ERROR(
    hipsparseCreateCsr(&csrMat.descr, static_cast<int64_t>(size), static_cast<int64_t>(size), static_cast<int64_t>(nnz), reinterpret_cast<void *>(csrMat.rowOffsetsPtr), reinterpret_cast<void *>(csrMat.colIndPtr), reinterpret_cast<void *>(csrMat.valuesPtr), HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, cuTraits<T>::valueType));
}

template <typename T>
void cuFreeMod(T *&ptr)
{
  if (ptr != nullptr) {
    CHECK_CUDA_ERROR(hipFree(ptr));
    ptr = nullptr;
  }
}

void cublasDot(hipblasHandle_t handle, int n, const float *x, const float *y, float *result)
{
  CHECK_CUDA_ERROR(hipblasSdot(handle, n, x, 1, y, 1, result));
}

void cublasDot(hipblasHandle_t handle, int n, const double *x, const double *y, double *result)
{
  CHECK_CUDA_ERROR(hipblasDdot(handle, n, x, 1, y, 1, result));
}

void cublasNorm(hipblasHandle_t handle, int n, const float *x, float *result)
{
  CHECK_CUDA_ERROR(hipblasSnrm2(handle, n, x, 1, result));
}

void cublasNorm(hipblasHandle_t handle, int n, const double *x, double *result)
{
  CHECK_CUDA_ERROR(hipblasDnrm2(handle, n, x, 1, result));
}

void cublasAXPY(hipblasHandle_t handle, int n, const float *alpha, const float *x, float *y)
{
  CHECK_CUDA_ERROR(hipblasSaxpy(handle, n, alpha, x, 1, y, 1));
}

void cublasAXPY(hipblasHandle_t handle, int n, const double *alpha, const double *x, double *y)
{
  CHECK_CUDA_ERROR(hipblasDaxpy(handle, n, alpha, x, 1, y, 1));
}

void cublasScal(hipblasHandle_t handle, int n, const float *alpha, float *x)
{
  CHECK_CUDA_ERROR(hipblasSscal(handle, n, alpha, x, 1));
}

void cublasScal(hipblasHandle_t handle, int n, const double *alpha, double *x)
{
  CHECK_CUDA_ERROR(hipblasDscal(handle, n, alpha, x, 1));
}

template <typename T>
void cufctSolver<T>::solve(T *u, const T *b, int maxIter, T rtol, T atol)
{
  if (dlPtr == nullptr || dPtr == nullptr || duPtr == nullptr || csrMat.descr == nullptr) {
    std::cerr << "The internal data have not been initialized!\n";
    std::cerr << "There will be nothing to do in this routine.\n";
    return;
  }

  size_t size = dims[0] * dims[1] * dims[2];
  /* Malloc and copy u. */
  dnVec<T> u_d{nullptr, nullptr};
  CHECK_CUDA_ERROR(hipMalloc(reinterpret_cast<void **>(&u_d.ptr), size * sizeof(T)));
  CHECK_CUDA_ERROR(hipMemcpy(reinterpret_cast<void *>(u_d.ptr), reinterpret_cast<void *>(&u[0]), size * sizeof(T), hipMemcpyHostToDevice));
  CHECK_CUDA_ERROR(hipsparseCreateDnVec(&u_d.descr, static_cast<int64_t>(size), reinterpret_cast<void *>(u_d.ptr), cuTraits<T>::valueType));

  /* Malloc r, r <= b, r <- r - Au_d */
  dnVec<T> r{nullptr, nullptr};
  CHECK_CUDA_ERROR(hipMalloc(reinterpret_cast<void **>(&r.ptr), size * sizeof(T)));
  CHECK_CUDA_ERROR(hipMemcpy(reinterpret_cast<void *>(r.ptr), &b[0], size * sizeof(T), hipMemcpyHostToDevice));
  CHECK_CUDA_ERROR(hipsparseCreateDnVec(&r.descr, static_cast<int64_t>(size), reinterpret_cast<void *>(r.ptr), cuTraits<T>::valueType));
  T bNorm;
  cublasNorm(blasHandle, size, &r.ptr[0], &bNorm);
  size_t bufferMVSize;
  T      alpha{static_cast<T>(-1)}, beta{static_cast<T>(1)};
  CHECK_CUDA_ERROR(hipsparseSpMV_bufferSize(sprHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, csrMat.descr, u_d.descr, &beta, r.descr, cuTraits<T>::valueType, HIPSPARSE_SPMV_ALG_DEFAULT, &bufferMVSize));
  void *bufferMV;
  CHECK_CUDA_ERROR(hipMalloc(&bufferMV, bufferMVSize));
  CHECK_CUDA_ERROR(hipsparseSpMV(sprHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, csrMat.descr, u_d.descr, &beta, r.descr, cuTraits<T>::valueType, HIPSPARSE_SPMV_ALG_DEFAULT, bufferMV));

  /* Malloc z, z <= r, z <- inv(M) z */
  dnVec<T> z{nullptr, nullptr};
  CHECK_CUDA_ERROR(hipMalloc(reinterpret_cast<void **>(&z.ptr), size * sizeof(T)));
  CHECK_CUDA_ERROR(hipMemcpy(reinterpret_cast<void *>(z.ptr), reinterpret_cast<void *>(&r.ptr[0]), size * sizeof(T), hipMemcpyDeviceToDevice));
  CHECK_CUDA_ERROR(hipsparseCreateDnVec(&z.descr, static_cast<int64_t>(size), reinterpret_cast<void *>(z.ptr), cuTraits<T>::valueType));
  precondSolver(&z.ptr[0]);

  /* Malloc p, p <= z */
  dnVec<T> p{nullptr, nullptr};
  CHECK_CUDA_ERROR(hipMalloc(reinterpret_cast<void **>(&p.ptr), size * sizeof(T)));
  CHECK_CUDA_ERROR(hipMemcpy(reinterpret_cast<void *>(p.ptr), reinterpret_cast<void *>(&z.ptr[0]), size * sizeof(T), hipMemcpyDeviceToDevice));
  CHECK_CUDA_ERROR(hipsparseCreateDnVec(&p.descr, static_cast<int64_t>(size), reinterpret_cast<void *>(p.ptr), cuTraits<T>::valueType));

  /* Malloc aux */
  dnVec<T> aux{nullptr, nullptr};
  CHECK_CUDA_ERROR(hipMalloc(reinterpret_cast<void **>(&aux.ptr), size * sizeof(T)));
  CHECK_CUDA_ERROR(hipsparseCreateDnVec(&aux.descr, static_cast<int64_t>(size), reinterpret_cast<void *>(aux.ptr), cuTraits<T>::valueType));

  T rDz, rDzNew, rNorm;
  cublasDot(blasHandle, size, &r.ptr[0], &z.ptr[0], &rDz);
  for (int itrIdx{0}; itrIdx < maxIter; ++itrIdx) {
    /* aux <- A p + 0*aux */
    alpha = static_cast<T>(1);
    beta  = static_cast<T>(0);
    CHECK_CUDA_ERROR(hipsparseSpMV(sprHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, csrMat.descr, p.descr, &beta, aux.descr, cuTraits<T>::valueType, HIPSPARSE_SPMV_ALG_DEFAULT, bufferMV));

    /* beta <- p (dot) aux, alpha <- rDz / beta */
    cublasDot(blasHandle, size, &p.ptr[0], &aux.ptr[0], &beta);
    alpha = rDz / beta;

    /* u <- alpha p + u, r <- -alpha p + r */
    cublasAXPY(blasHandle, size, &alpha, &p.ptr[0], &u_d.ptr[0]);
    alpha *= -1;
    cublasAXPY(blasHandle, size, &alpha, &p.ptr[0], &r.ptr[0]);

    /* Check convergence reasons. */
    cublasNorm(blasHandle, size, &r.ptr[0], &rNorm);
    if (rNorm <= bNorm * rtol) {
      std::printf("Reach rtol=%.6e, the solver exits with residual=%.6e and iterations=%d.\n", rtol, rNorm, itrIdx + 1);
      break;
    }
    if (rNorm <= atol) {
      std::printf("Reach atol=%.6e, the solver exits with residual=%.6e and iterations=%d.\n", atol, rNorm, itrIdx + 1);
      break;
    }
    if (maxIter - 1 == itrIdx) {
      std::printf("Reach maxIter=%d, the solver exits with residual=%.6e and iterations=%d.\n", maxIter, rNorm, itrIdx + 1);
      break;
    }
#ifdef DEBUG
    std::printf("itrIdx=%d,\tresidual=%.6e.\n", itrIdx, rNorm);
#endif

    /* z <= r, z <- inv(M) z */
    CHECK_CUDA_ERROR(hipMemcpy(&z.ptr[0], &r.ptr[0], size * sizeof(T), hipMemcpyDeviceToDevice));
    precondSolver(&z.ptr[0]);

    /* rDzNew <- r (dot) z, beta <- rDzNew / rDz */
    cublasDot(blasHandle, size, &r.ptr[0], &z.ptr[0], &rDzNew);
    beta = rDzNew / rDz;

    /* p <- beta p, p <- z + p */
    cublasScal(blasHandle, size, &beta, &p.ptr[0]);
    alpha = static_cast<T>(1);
    cublasAXPY(blasHandle, size, &alpha, &z.ptr[0], &p.ptr[0]);

    /* rDz <- rDzNew */
    rDz = rDzNew;
  }

  /* Copy u_d back to u. */
  CHECK_CUDA_ERROR(hipMemcpy(reinterpret_cast<void *>(&u[0]), reinterpret_cast<void *>(&u_d.ptr[0]), size * sizeof(T), hipMemcpyDeviceToHost));

  /* Free all resources. */
  CHECK_CUDA_ERROR(hipsparseDestroyDnVec(aux.descr));
  cuFreeMod(aux.ptr);

  CHECK_CUDA_ERROR(hipsparseDestroyDnVec(p.descr));
  cuFreeMod(p.ptr);

  CHECK_CUDA_ERROR(hipsparseDestroyDnVec(z.descr));
  cuFreeMod(z.ptr);

  cuFreeMod(bufferMV);

  CHECK_CUDA_ERROR(hipsparseDestroyDnVec(r.descr));
  cuFreeMod(r.ptr);

  CHECK_CUDA_ERROR(hipsparseDestroyDnVec(u_d.descr));
  cuFreeMod(u_d.ptr);
}

template <typename T>
void cufctSolver<T>::solveWithoutPrecond(T *u, const T *b, int maxIter, T rtol, T atol)
{
  if (csrMat.descr == nullptr) {
    std::cerr << "The internal data have not been initialized!\n";
    std::cerr << "There will be nothing to do in this routine.\n";
    return;
  }

  size_t size = dims[0] * dims[1] * dims[2];
  /* Malloc and copy u. */
  dnVec<T> u_d{nullptr, nullptr};
  CHECK_CUDA_ERROR(hipMalloc(reinterpret_cast<void **>(&u_d.ptr), size * sizeof(T)));
  CHECK_CUDA_ERROR(hipMemcpy(reinterpret_cast<void *>(u_d.ptr), reinterpret_cast<void *>(&u[0]), size * sizeof(T), hipMemcpyHostToDevice));
  CHECK_CUDA_ERROR(hipsparseCreateDnVec(&u_d.descr, static_cast<int64_t>(size), reinterpret_cast<void *>(u_d.ptr), cuTraits<T>::valueType));

  /* Malloc r, r <= b, r <- r - Au_d */
  dnVec<T> r{nullptr, nullptr};
  CHECK_CUDA_ERROR(hipMalloc(reinterpret_cast<void **>(&r.ptr), size * sizeof(T)));
  CHECK_CUDA_ERROR(hipMemcpy(reinterpret_cast<void *>(r.ptr), &b[0], size * sizeof(T), hipMemcpyHostToDevice));
  CHECK_CUDA_ERROR(hipsparseCreateDnVec(&r.descr, static_cast<int64_t>(size), reinterpret_cast<void *>(r.ptr), cuTraits<T>::valueType));
  T bNorm;
  cublasNorm(blasHandle, size, &r.ptr[0], &bNorm);
  size_t bufferMVSize;
  T      alpha{static_cast<T>(-1)}, beta{static_cast<T>(1)};
  CHECK_CUDA_ERROR(hipsparseSpMV_bufferSize(sprHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, csrMat.descr, u_d.descr, &beta, r.descr, cuTraits<T>::valueType, HIPSPARSE_SPMV_ALG_DEFAULT, &bufferMVSize));
  void *bufferMV;
  CHECK_CUDA_ERROR(hipMalloc(&bufferMV, bufferMVSize));
  CHECK_CUDA_ERROR(hipsparseSpMV(sprHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, csrMat.descr, u_d.descr, &beta, r.descr, cuTraits<T>::valueType, HIPSPARSE_SPMV_ALG_DEFAULT, bufferMV));

  /* Malloc p, p <= z */
  dnVec<T> p{nullptr, nullptr};
  CHECK_CUDA_ERROR(hipMalloc(reinterpret_cast<void **>(&p.ptr), size * sizeof(T)));
  CHECK_CUDA_ERROR(hipMemcpy(reinterpret_cast<void *>(p.ptr), reinterpret_cast<void *>(&r.ptr[0]), size * sizeof(T), hipMemcpyDeviceToDevice));
  CHECK_CUDA_ERROR(hipsparseCreateDnVec(&p.descr, static_cast<int64_t>(size), reinterpret_cast<void *>(p.ptr), cuTraits<T>::valueType));

  /* Malloc aux */
  dnVec<T> aux{nullptr, nullptr};
  CHECK_CUDA_ERROR(hipMalloc(reinterpret_cast<void **>(&aux.ptr), size * sizeof(T)));
  CHECK_CUDA_ERROR(hipsparseCreateDnVec(&aux.descr, static_cast<int64_t>(size), reinterpret_cast<void *>(aux.ptr), cuTraits<T>::valueType));

  T rDr, rDrNew, rNorm;
  cublasDot(blasHandle, size, &r.ptr[0], &r.ptr[0], &rDr);

  for (int itrIdx{0}; itrIdx < maxIter; ++itrIdx) {
    /* aux <- A p + 0*aux */
    alpha = static_cast<T>(1);
    beta  = static_cast<T>(0);
    CHECK_CUDA_ERROR(hipsparseSpMV(sprHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, csrMat.descr, p.descr, &beta, aux.descr, cuTraits<T>::valueType, HIPSPARSE_SPMV_ALG_DEFAULT, bufferMV));

    /* beta <- p (dot) aux, alpha <- rDr / beta */
    cublasDot(blasHandle, size, &p.ptr[0], &aux.ptr[0], &beta);
    alpha = rDr / beta;

    /* u <- alpha p + u, r <- -alpha p + r */
    cublasAXPY(blasHandle, size, &alpha, &p.ptr[0], &u_d.ptr[0]);
    alpha *= -1;
    cublasAXPY(blasHandle, size, &alpha, &p.ptr[0], &r.ptr[0]);

    /* Check convergence reasons. */
    rNorm = std::sqrt(rDr);
    if (rNorm <= bNorm * rtol) {
      std::printf("Reach rtol=%.6e, the solver exits with residual=%.6e and iterations=%d.\n", rtol, rNorm, itrIdx + 1);
      break;
    }
    if (rNorm <= atol) {
      std::printf("Reach atol=%.6e, the solver exits with residual=%.6e and iterations=%d.\n", atol, rNorm, itrIdx + 1);
      break;
    }
    if (maxIter - 1 == itrIdx) {
      std::printf("Reach maxIter=%d, the solver exits with residual=%.6e and iterations=%d.\n", maxIter, rNorm, itrIdx + 1);
      break;
    }
#ifdef DEBUG
    std::printf("itrIdx=%d,\tresidual=%.6e.\n", itrIdx, rNorm);
#endif

    /* rDrNew <- r (dot) r, beta <- rDrNew / rDr */
    cublasDot(blasHandle, size, &r.ptr[0], &r.ptr[0], &rDrNew);
    beta = rDrNew / rDr;

    /* p <- beta p, p <- r + p */
    cublasScal(blasHandle, size, &beta, &p.ptr[0]);
    alpha = static_cast<T>(1);
    cublasAXPY(blasHandle, size, &alpha, &r.ptr[0], &p.ptr[0]);

    /* rDr <- rDrNew */
    rDr = rDrNew;
  }
  /* Copy u_d back to u. */
  CHECK_CUDA_ERROR(hipMemcpy(reinterpret_cast<void *>(&u[0]), reinterpret_cast<void *>(&u_d.ptr[0]), size * sizeof(T), hipMemcpyDeviceToHost));

  /* Free all resources. */
  CHECK_CUDA_ERROR(hipsparseDestroyDnVec(aux.descr));
  cuFreeMod(aux.ptr);

  CHECK_CUDA_ERROR(hipsparseDestroyDnVec(p.descr));
  cuFreeMod(p.ptr);

  cuFreeMod(bufferMV);

  CHECK_CUDA_ERROR(hipsparseDestroyDnVec(r.descr));
  cuFreeMod(r.ptr);

  CHECK_CUDA_ERROR(hipsparseDestroyDnVec(u_d.descr));
  cuFreeMod(u_d.ptr);
}

template <typename T>
cufctSolver<T>::~cufctSolver()
{
  CHECK_CUDA_ERROR(hipblasDestroy(blasHandle));
  blasHandle = nullptr;

  if (csrMat.descr != nullptr) {
    CHECK_CUDA_ERROR(hipsparseDestroySpMat(csrMat.descr));
    csrMat.descr = nullptr;
  }

  cuFreeMod(csrMat.valuesPtr);
  cuFreeMod(csrMat.colIndPtr);
  cuFreeMod(csrMat.rowOffsetsPtr);
  cuFreeMod(tridSolverBuffer);
  cuFreeMod(duPtr);
  cuFreeMod(dPtr);
  cuFreeMod(dlPtr);

  CHECK_CUDA_ERROR(hipsparseDestroy(sprHandle));
  sprHandle = nullptr;

  CHECK_CUDA_ERROR(hipfftDestroy(c2rPlan));
  CHECK_CUDA_ERROR(hipfftDestroy(r2cPlan));

  cuFreeMod(compBuffer);
  cuFreeMod(realBuffer);
}

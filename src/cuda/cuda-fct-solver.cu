#include "hip/hip_runtime.h"
#include "cuda-fct-solver.hpp"

#define MAX_THREADS_PER_BLOCK  1024
#define WARP_SIZE              32
#define FCT_POST_STENCIL_WIDTH 2
#define IFCT_PRE_STENCIL_WIDTH 4

void check(hipError_t status, char const *const func, char const *const file, int const line)
{
  if (status != hipSuccess) {
    std::cerr << "CUDA Runtime Error at: " << file << ":" << line << std::endl;
    std::cerr << hipGetErrorString(status) << " " << func << std::endl;
    std::exit(EXIT_FAILURE);
  }
}

void check(hipfftResult status, char const *const func, char const *const file, int const line)
{
  if (status != HIPFFT_SUCCESS) {
    std::cerr << "cufft Error at: " << file << ":" << line << std::endl;
    std::cerr << "with code (" << status << ") " << func << std::endl;
    std::exit(EXIT_FAILURE);
  }
}

void check(hipsparseStatus_t status, char const *const func, char const *const file, int const line)
{
  if (status != HIPSPARSE_STATUS_SUCCESS) {
    std::cerr << "cusparse Error at: " << file << ":" << line << std::endl;
    std::cerr << hipsparseGetErrorString(status) << " " << func << std::endl;
    std::exit(EXIT_FAILURE);
  }
}

void checkLast(char const *const file, int const line)
{
  hipError_t err{hipGetLastError()};
  if (err != hipSuccess) {
    std::cerr << "CUDA Runtime Error at: " << file << ":" << line << std::endl;
    std::cerr << hipGetErrorString(err) << std::endl;
    std::exit(EXIT_FAILURE);
  }
}

__device__ int getIdxFrom3dIdx_d(const int i, const int j, const int k, const int N, const int P)
{
  return i * N * P + (j * P) + k;
}

__device__ int getIdxFrom3dIdxHalf(const int i, const int j, const int k, const int N, const int P)
{
  return i * (N / 2 + 1) * P + (j * P) + k;
}

/*
    Note that P may not be a 32x integer, which my cause warp divergences.
    Hence this routine is designed to make every 32 threads operate the
    same i and j.
*/

__host__ __device__ int getPmod(const int P)
{
  return (P / WARP_SIZE + 1) * WARP_SIZE;
}

__device__ void get3dIdxFromThreadIdx(int &i, int &j, int &k, const int glbThreadIdx, const int N, const int P, const int Pmod)
{
  i = glbThreadIdx / (N * Pmod);
  j = (glbThreadIdx / Pmod) % N;
  k = (glbThreadIdx % Pmod) % P;
}

__device__ hipComplex getExpItheta(const float theta)
{
  return make_hipComplex(cosf(theta), sinf(theta));
}

__device__ hipDoubleComplex getExpItheta(const double theta)
{
  return make_hipDoubleComplex(cos(theta), sin(theta));
}

__device__ hipComplex hipCmul(hipComplex cVar1, hipComplex cVar2)
{
  return hipCmulf(cVar1, cVar2);
}

template <typename T>
__global__ void fctPre(T *out, const T *in, const int M, const int N, const int P)
{
  size_t       glbThreadIdx{blockIdx.x * blockDim.x + threadIdx.x};
  int          i{0}, j{0}, k{0};
  int          Pmod{getPmod(P)};
  __shared__ T in_buffer[MAX_THREADS_PER_BLOCK];

  if (glbThreadIdx < M * N * Pmod) {
    int idx_req{0};
    get3dIdxFromThreadIdx(i, j, k, glbThreadIdx, N, P, Pmod);

    if (i < (M + 1) / 2 && j < (N + 1) / 2) idx_req = getIdxFrom3dIdx_d(2 * i, 2 * j, k, N, P);
    if ((M + 1) / 2 <= i && j < (N + 1) / 2) idx_req = getIdxFrom3dIdx_d(2 * M - 2 * i - 1, 2 * j, k, N, P);
    if (i < (M + 1) / 2 && (N + 1) / 2 <= j) idx_req = getIdxFrom3dIdx_d(2 * i, 2 * N - 2 * j - 1, k, N, P);
    if ((M + 1) / 2 <= i && (N + 1) / 2 <= j) idx_req = getIdxFrom3dIdx_d(2 * M - 2 * i - 1, 2 * N - 2 * j - 1, k, N, P);

    in_buffer[threadIdx.x] = in[idx_req];
  }
  __syncthreads();

  if (glbThreadIdx < M * N * Pmod) {
    // get3dIdxFromThreadIdx(i, j, k, glbThreadIdx, N, P, Pmod);
    int idx_tar{getIdxFrom3dIdx_d(i, j, k, N, P)};
    out[idx_tar] = in_buffer[threadIdx.x];
  }
}

template <typename T>
__global__ void fctPost(T *out_hat, const decltype(cuTraits<T>::compVar) *in_hat, const int M, const int N, const int P)
{
  using complex_T = decltype(cuTraits<T>::compVar);
  size_t       glbThreadIdx{blockIdx.x * blockDim.x + threadIdx.x};
  int          i_p{0}, j_p{0}, k{0};
  int          Pmod{getPmod(P)};
  __shared__ T in_hat_buffer[2 * FCT_POST_STENCIL_WIDTH][MAX_THREADS_PER_BLOCK + 1];
  // Cannot use cuda::std::complex<T> here.
  // Avoid bank conflicts, we add a padding to every row here.

  if (glbThreadIdx < M * N * Pmod) {
    get3dIdxFromThreadIdx(i_p, j_p, k, glbThreadIdx, N, P, Pmod);
    int idx_req{0};
    if (1 <= i_p && j_p < N / 2 + 1) {
      idx_req                       = getIdxFrom3dIdxHalf(i_p, j_p, k, N, P);
      in_hat_buffer[0][threadIdx.x] = in_hat[idx_req].x;
      in_hat_buffer[1][threadIdx.x] = in_hat[idx_req].y;

      idx_req                       = getIdxFrom3dIdxHalf(M - i_p, j_p, k, N, P);
      in_hat_buffer[2][threadIdx.x] = in_hat[idx_req].x;
      in_hat_buffer[3][threadIdx.x] = in_hat[idx_req].y;
    }
    if (0 == i_p && j_p < N / 2 + 1) {
      idx_req                       = getIdxFrom3dIdxHalf(0, j_p, k, N, P);
      in_hat_buffer[0][threadIdx.x] = in_hat[idx_req].x;
      in_hat_buffer[1][threadIdx.x] = in_hat[idx_req].y;

      idx_req                       = getIdxFrom3dIdxHalf(0, j_p, k, N, P);
      in_hat_buffer[2][threadIdx.x] = in_hat[idx_req].x;
      in_hat_buffer[3][threadIdx.x] = in_hat[idx_req].y;
    }
    if (1 <= i_p && N / 2 + 1 <= j_p) {
      idx_req                       = getIdxFrom3dIdxHalf(M - i_p, N - j_p, k, N, P);
      in_hat_buffer[0][threadIdx.x] = in_hat[idx_req].x;
      in_hat_buffer[1][threadIdx.x] = in_hat[idx_req].y;

      idx_req                       = getIdxFrom3dIdxHalf(i_p, N - j_p, k, N, P);
      in_hat_buffer[2][threadIdx.x] = in_hat[idx_req].x;
      in_hat_buffer[3][threadIdx.x] = in_hat[idx_req].y;
    }
    if (0 == i_p && N / 2 + 1 <= j_p) {
      idx_req                       = getIdxFrom3dIdxHalf(0, N - j_p, k, N, P);
      in_hat_buffer[0][threadIdx.x] = in_hat[idx_req].x;
      in_hat_buffer[1][threadIdx.x] = in_hat[idx_req].y;

      idx_req                       = getIdxFrom3dIdxHalf(0, N - j_p, k, N, P);
      in_hat_buffer[2][threadIdx.x] = in_hat[idx_req].x;
      in_hat_buffer[3][threadIdx.x] = in_hat[idx_req].y;
    }
  }
  __syncthreads();

  if (glbThreadIdx < M * N * Pmod) {
    // get3dIdxFromThreadIdx(i_p, j_p, k, glbThreadIdx, N, P, Pmod);
    int       idx_tar{getIdxFrom3dIdx_d(i_p, j_p, k, N, P)};
    T         i_theta, j_theta, cuPi{static_cast<T>(M_PI)}, temp0, temp1;
    complex_T ninj_exp, nipj_exp, tempBuff0, tempBuff1;
    i_theta  = static_cast<T>(i_p) / static_cast<T>(2 * M) * cuPi;
    j_theta  = static_cast<T>(j_p) / static_cast<T>(2 * N) * cuPi;
    ninj_exp = getExpItheta(-i_theta - j_theta);
    nipj_exp = getExpItheta(-i_theta + j_theta);

    if (1 <= j_p && j_p < N / 2 + 1) {
      tempBuff0.x      = in_hat_buffer[0][threadIdx.x];
      tempBuff0.y      = in_hat_buffer[1][threadIdx.x];
      temp0            = ninj_exp.x * tempBuff0.x - ninj_exp.y * tempBuff0.y;
      tempBuff1.x      = in_hat_buffer[2][threadIdx.x];
      tempBuff1.y      = -in_hat_buffer[3][threadIdx.x];
      temp1            = nipj_exp.x * tempBuff1.x - nipj_exp.y * tempBuff1.y;
      out_hat[idx_tar] = (temp0 + temp1) / 2;
      return;
    }
    if (N / 2 + 1 <= j_p) {
      tempBuff0.x      = in_hat_buffer[0][threadIdx.x];
      tempBuff0.y      = -in_hat_buffer[1][threadIdx.x];
      temp0            = ninj_exp.x * tempBuff0.x - ninj_exp.y * tempBuff0.y;
      tempBuff1.x      = in_hat_buffer[2][threadIdx.x];
      tempBuff1.y      = in_hat_buffer[3][threadIdx.x];
      temp1            = nipj_exp.x * tempBuff1.x - nipj_exp.y * tempBuff1.y;
      out_hat[idx_tar] = (temp0 + temp1) / 2;
      return;
    }
    if (0 == j_p) {
      tempBuff0.x      = in_hat_buffer[0][threadIdx.x];
      tempBuff0.y      = in_hat_buffer[1][threadIdx.x];
      out_hat[idx_tar] = ninj_exp.x * tempBuff0.x - ninj_exp.y * tempBuff0.y;
      return;
    }
  } else return;
}

template <typename T>
__global__ void ifctPre(decltype(cuTraits<T>::compVar) *out_hat, const T *in_hat, const int M, const int N, const int P)
{
  using complex_T = decltype(cuTraits<T>::compVar);
  size_t       glbThreadIdx{blockIdx.x * blockDim.x + threadIdx.x};
  int          i_p{0}, j_p{0}, k{0};
  int          Pmod{(P / WARP_SIZE + 1) * WARP_SIZE};
  __shared__ T in_hat_buffer[IFCT_PRE_STENCIL_WIDTH][MAX_THREADS_PER_BLOCK + 1];
  /* Avoid bank conflicts, we add a pad to every row here. */

  if (glbThreadIdx < M * N * Pmod) {
    get3dIdxFromThreadIdx(i_p, j_p, k, glbThreadIdx, N, P, Pmod);
    int idx_req{getIdxFrom3dIdx_d(i_p, j_p, k, N, P)};
    in_hat_buffer[0][threadIdx.x] = in_hat[idx_req];
    if (0 < i_p && 0 < j_p) {
      idx_req                       = getIdxFrom3dIdx_d(M - i_p, N - j_p, k, N, P);
      in_hat_buffer[1][threadIdx.x] = in_hat[idx_req];

      idx_req                       = getIdxFrom3dIdx_d(M - i_p, j_p, k, N, P);
      in_hat_buffer[2][threadIdx.x] = in_hat[idx_req];

      idx_req                       = getIdxFrom3dIdx_d(i_p, N - j_p, k, N, P);
      in_hat_buffer[3][threadIdx.x] = in_hat[idx_req];
    }
    if (0 == i_p && 0 < j_p) {
      in_hat_buffer[1][threadIdx.x] = 0;

      in_hat_buffer[2][threadIdx.x] = 0;

      idx_req                       = getIdxFrom3dIdx_d(0, N - j_p, k, N, P);
      in_hat_buffer[3][threadIdx.x] = in_hat[idx_req];
    }
    if (0 < i_p && 0 == j_p) {
      in_hat_buffer[1][threadIdx.x] = 0;

      idx_req                       = getIdxFrom3dIdx_d(M - i_p, 0, k, N, P);
      in_hat_buffer[2][threadIdx.x] = in_hat[idx_req];

      in_hat_buffer[3][threadIdx.x] = 0;
    }
    if (0 == i_p && 0 == j_p) {
      in_hat_buffer[1][threadIdx.x] = 0;

      in_hat_buffer[2][threadIdx.x] = 0;

      in_hat_buffer[3][threadIdx.x] = 0;
    }
  }
  __syncthreads();

  if (glbThreadIdx < M * N * Pmod && j_p < N / 2 + 1) {
    T         i_theta, j_theta, cuPi{static_cast<T>(M_PI)};
    complex_T temp, pipj_exp;
    i_theta  = static_cast<T>(i_p) / static_cast<T>(2 * M) * cuPi;
    j_theta  = static_cast<T>(j_p) / static_cast<T>(2 * N) * cuPi;
    pipj_exp = getExpItheta(i_theta + j_theta);
    temp.x   = in_hat_buffer[0][threadIdx.x] - in_hat_buffer[1][threadIdx.x];
    temp.y   = -(in_hat_buffer[2][threadIdx.x] + in_hat_buffer[3][threadIdx.x]);
    int idx_tar{getIdxFrom3dIdxHalf(i_p, j_p, k, N, P)};
    out_hat[idx_tar] = hipCmul(pipj_exp, temp);
    return;
  } else return;
}

template <typename T>
__global__ void ifctPost(T *out, const T *in, const int M, const int N, const int P)
{
  size_t       glbThreadIdx{blockIdx.x * blockDim.x + threadIdx.x};
  int          i{0}, j{0}, k{0};
  int          Pmod{(P / WARP_SIZE + 1) * WARP_SIZE};
  __shared__ T in_buffer[MAX_THREADS_PER_BLOCK];

  if (glbThreadIdx < M * N * Pmod) {
    get3dIdxFromThreadIdx(i, j, k, glbThreadIdx, N, P, Pmod);
    int idx_req{0};
    if (0 == i % 2 && 0 == j % 2) idx_req = getIdxFrom3dIdx_d(i / 2, j / 2, k, N, P);
    if (0 == i % 2 && 1 == j % 2) idx_req = getIdxFrom3dIdx_d(i / 2, N - (j + 1) / 2, k, N, P);
    if (1 == i % 2 && 0 == j % 2) idx_req = getIdxFrom3dIdx_d(M - (i + 1) / 2, j / 2, k, N, P);
    if (1 == i % 2 && 1 == j % 2) idx_req = getIdxFrom3dIdx_d(M - (i + 1) / 2, N - (j + 1) / 2, k, N, P);

    in_buffer[threadIdx.x] = in[idx_req];
  }
  __syncthreads();

  if (glbThreadIdx < M * N * Pmod) {
    int     idx_tar{getIdxFrom3dIdx_d(i, j, k, N, P)};
    const T scalFactor{static_cast<T>(1) / (M * N)};
    /* cuFFT performs un-normalized FFTs! */
    out[idx_tar] = in_buffer[threadIdx.x] * scalFactor;
  }
}

template <typename T>
cufctSolver<T>::cufctSolver(const int _M, const int _N, const int _P) : dims{_M, _N, _P}, realBuffer{nullptr}, compBuffer{nullptr}, dlPtr{nullptr}, dPtr{nullptr}, duPtr{nullptr}, tridSolverBuffer{nullptr}
{
  CHECK_CUDA_ERROR(hipMalloc(reinterpret_cast<void **>(&realBuffer), sizeof(T) * _M * _N * _P));
  CHECK_CUDA_ERROR(hipMalloc(reinterpret_cast<void **>(&compBuffer), sizeof(cuCompType) * _M * (_N / 2 + 1) * _P));
  // CHECK_CUDA_ERROR(hipMalloc(reinterpret_cast<void **>(&compBuffer), sizeof(cuCompType) * _M * _N * _P));
  /* Due to the C2R transformation, we could use less memory for complex vectors. */
  /* Works on the cufft context. */
  int dimsHalf[3]{_M, _N / 2 + 1, _P};

  CHECK_CUDA_ERROR(hipfftCreate(&r2cPlan));
  CHECK_CUDA_ERROR(hipfftPlanMany(&r2cPlan, 2, &dims[0], &dims[0], _P, 1, &dimsHalf[0], _P, 1, cuTraits<T>::r2cType, _P));
  /* It is strange that cufft does not explain why need dimsHalf here. */
  /* There are some mismatch between fftw and cufft. */
  CHECK_CUDA_ERROR(hipfftCreate(&c2rPlan));
  CHECK_CUDA_ERROR(hipfftPlanMany(&c2rPlan, 2, &dims[0], &dimsHalf[0], _P, 1, &dims[0], _P, 1, cuTraits<T>::c2rType, _P));
  // CHECK_CUDA_ERROR(hipfftCreate(&c2cPlan));
  // CHECK_CUDA_ERROR(hipfftPlanMany(&c2rPlan, 2, &dims[0], &dims[0], dims[2], 1, &dims[0], dims[2], 1, cuTraits<T>::c2cType, dims[2]));

  /* Works on the cusparse context. */
  CHECK_CUDA_ERROR(hipsparseCreate(&cusprHandle));
}

hipfftResult cufftReal2Comp(hipfftHandle plan, float *idata, hipComplex *odata)
{
  return hipfftExecR2C(plan, reinterpret_cast<hipfftReal *>(idata), reinterpret_cast<hipfftComplex *>(odata));
}

hipfftResult cufftReal2Comp(hipfftHandle plan, double *idata, hipDoubleComplex *odata)
{
  return hipfftExecD2Z(plan, reinterpret_cast<hipfftDoubleReal *>(idata), reinterpret_cast<hipfftDoubleComplex *>(odata));
}

template <typename T>
void viewRealVec(std::vector<T> &vec)
{
  for (int i{0}; i < vec.size(); ++i) std::cout << '[' << i << "]=" << vec[i] << ' ';
  std::cout << '\n';
}

template <typename T>
void viewCompVec(std::vector<decltype(cuTraits<T>::compVar)> &vec)
{
  for (int i{0}; i < vec.size(); ++i) {
    std::cout << '[' << i << "]=" << vec[i].x;
    if (vec[i].y >= 0) std::cout << '+';
    std::cout << vec[i].y << "I\n";
  }
  std::cout << '\n';
}

template <typename T>
void cufctSolver<T>::fctForward(T *v)
{
  int M{dims[0]}, N{dims[1]}, P{dims[2]};
  int blockSize{0};   // The launch configurator returned block size
  int minGridSize{0}; // The minimum grid size needed to achieve the maximum occupancy for a full device launch
  int gridSize{0};    // The actual grid size needed, based on input size
  int Pmod{getPmod(P)};

  // std::vector<T>                              reViewer(M * N * P);
  // std::vector<decltype(cuTraits<T>::compVar)> cmViewer(M * (N / 2 + 1) * P);
  // std::cout << std::fixed << std::showpoint << std::setprecision(3);

  // CHECK_CUDA_ERROR(hipMemcpy(&reViewer[0], &v[0], M * N * P * sizeof(T), hipMemcpyDeviceToHost));
  // viewRealVec(reViewer);

  CHECK_CUDA_ERROR(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, &fctPre<T>, 0, 0));
  blockSize = (blockSize / WARP_SIZE) * WARP_SIZE; // This should be useless.
  if (blockSize < P) {
    std::cout << "Recommended blocksize=" << blockSize << " < P=" << P << ", reset blocksize=" << MAX_THREADS_PER_BLOCK << '\n';
    blockSize = MAX_THREADS_PER_BLOCK;
  }
  gridSize = (M * N * Pmod + blockSize - 1) / blockSize;
  fctPre<T><<<gridSize, blockSize>>>(&realBuffer[0], &v[0], M, N, P);
  CHECK_LAST_CUDA_ERROR();

  // CHECK_CUDA_ERROR(hipMemcpy(&reViewer[0], &realBuffer[0], M * N * P * sizeof(T), hipMemcpyDeviceToHost));
  // viewRealVec(reViewer);

  CHECK_CUDA_ERROR(cufftReal2Comp(r2cPlan, &realBuffer[0], &compBuffer[0]));

  // CHECK_CUDA_ERROR(hipMemcpy(&cmViewer[0], &compBuffer[0], M * (N / 2 + 1) * P * sizeof(cuCompType), hipMemcpyDeviceToHost));
  // viewCompVec<T>(cmViewer);

  CHECK_CUDA_ERROR(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, &fctPost<T>, 0, 0));
  blockSize = (blockSize / WARP_SIZE) * WARP_SIZE; // This should be useless.
  if (blockSize < P) {
    std::cout << "Recommended blocksize=" << blockSize << " < P=" << P << ", reset blocksize=" << MAX_THREADS_PER_BLOCK << '\n';
    blockSize = MAX_THREADS_PER_BLOCK;
  }
  gridSize = (M * N * Pmod + blockSize - 1) / blockSize;
  fctPost<T><<<gridSize, blockSize>>>(&v[0], &compBuffer[0], M, N, P);
  CHECK_LAST_CUDA_ERROR();
}

hipfftResult cufftComp2Real(hipfftHandle plan, hipComplex *idata, float *odata)
{
  return hipfftExecC2R(plan, reinterpret_cast<hipfftComplex *>(idata), reinterpret_cast<hipfftReal *>(odata));
}

hipfftResult cufftComp2Real(hipfftHandle plan, hipDoubleComplex *idata, double *odata)
{
  return hipfftExecZ2D(plan, reinterpret_cast<hipfftDoubleComplex *>(idata), reinterpret_cast<hipfftDoubleReal *>(odata));
}

template <typename T>
void cufctSolver<T>::fctBackward(T *v)
{
  int M{dims[0]}, N{dims[1]}, P{dims[2]};
  int blockSize{0};   // The launch configurator returned block size
  int minGridSize{0}; // The minimum grid size needed to achieve the maximum occupancy for a full device launch
  int gridSize{0};    // The actual grid size needed, based on input size
  int Pmod{(P / WARP_SIZE + 1) * WARP_SIZE};

  // std::vector<T>                              reViewer(M * N * P);
  // std::vector<decltype(cuTraits<T>::compVar)> cmViewer(M * (N / 2 + 1) * P);
  // std::cout << std::fixed << std::showpoint << std::setprecision(3);

  // CHECK_CUDA_ERROR(hipMemcpy(&reViewer[0], &v[0], M * N * P * sizeof(T), hipMemcpyDeviceToHost));
  // viewRealVec(reViewer);

  CHECK_CUDA_ERROR(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, &ifctPre<T>, 0, 0));
  blockSize = (blockSize / WARP_SIZE) * WARP_SIZE; // This should be useless.
  if (blockSize < P) {
    std::cout << "Recommended blocksize=" << blockSize << " < P=" << P << ", reset blocksize=" << MAX_THREADS_PER_BLOCK << '\n';
    blockSize = MAX_THREADS_PER_BLOCK;
  }
  gridSize = (M * N * Pmod + blockSize - 1) / blockSize;
  ifctPre<T><<<gridSize, blockSize>>>(&compBuffer[0], &v[0], M, N, P);
  CHECK_LAST_CUDA_ERROR();

  // CHECK_CUDA_ERROR(hipMemcpy(&cmViewer[0], &compBuffer[0], M * (N / 2 + 1) * P * sizeof(cuCompType), hipMemcpyDeviceToHost));
  // viewCompVec<T>(cmViewer);

  CHECK_CUDA_ERROR(cufftComp2Real(c2rPlan, &compBuffer[0], &realBuffer[0]));

  // CHECK_CUDA_ERROR(hipMemcpy(&reViewer[0], &realBuffer[0], M * N * P * sizeof(T), hipMemcpyDeviceToHost));
  // viewRealVec(reViewer);

  CHECK_CUDA_ERROR(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, &ifctPost<T>, 0, 0));
  blockSize = (blockSize / WARP_SIZE) * WARP_SIZE; // This should be useless.
  if (blockSize < P) {
    std::cout << "Recommended blocksize=" << blockSize << " < P=" << P << ", reset blocksize=" << MAX_THREADS_PER_BLOCK << '\n';
    blockSize = MAX_THREADS_PER_BLOCK;
  }
  gridSize = (M * N * Pmod + blockSize - 1) / blockSize;
  ifctPost<T><<<gridSize, blockSize>>>(&v[0], &realBuffer[0], M, N, P);
  CHECK_LAST_CUDA_ERROR();

  // CHECK_CUDA_ERROR(hipMemcpy(&reViewer[0], &v[0], M * N * P * sizeof(T), hipMemcpyDeviceToHost));
  // viewRealVec(reViewer);
}

void gtsv2StridedBatch_bufferSizeExt(hipsparseHandle_t handle, int m, const float *dl, const float *d, const float *du, const float *x, int batchCount, int batchStride, size_t *bufferSizeInBytes)
{
  CHECK_CUDA_ERROR(hipsparseSgtsv2StridedBatch_bufferSizeExt(handle, m, dl, d, du, x, batchCount, batchStride, bufferSizeInBytes));
}

void gtsv2StridedBatch_bufferSizeExt(hipsparseHandle_t handle, int m, const double *dl, const double *d, const double *du, const double *x, int batchCount, int batchStride, size_t *bufferSizeInBytes)
{
  CHECK_CUDA_ERROR(hipsparseDgtsv2StridedBatch_bufferSizeExt(handle, m, dl, d, du, x, batchCount, batchStride, bufferSizeInBytes));
}

template <typename T>
void cufctSolver<T>::setTridSolverData(T *dl, T *d, T *du)
{
  if (dlPtr != nullptr || dPtr != nullptr || duPtr != nullptr) std::cerr << "The internal data have been initialized, be careful!\n";
  size_t size = dims[0] * dims[1] * dims[2];

  CHECK_CUDA_ERROR(hipMalloc(reinterpret_cast<void **>(&dlPtr), size * sizeof(T)));
  CHECK_CUDA_ERROR(hipMemcpy(reinterpret_cast<void *>(dlPtr), reinterpret_cast<void *>(dl), size * sizeof(T), hipMemcpyHostToDevice));

  CHECK_CUDA_ERROR(hipMalloc(reinterpret_cast<void **>(&dPtr), size * sizeof(T)));
  CHECK_CUDA_ERROR(hipMemcpy(reinterpret_cast<void *>(dPtr), reinterpret_cast<void *>(d), size * sizeof(T), hipMemcpyHostToDevice));

  CHECK_CUDA_ERROR(hipMalloc(reinterpret_cast<void **>(&duPtr), size * sizeof(T)));
  CHECK_CUDA_ERROR(hipMemcpy(reinterpret_cast<void *>(duPtr), reinterpret_cast<void *>(du), size * sizeof(T), hipMemcpyHostToDevice));

  size_t bufferSizeInBytes{0};
  int    M{dims[0]}, N{dims[1]}, P{dims[2]};
  gtsv2StridedBatch_bufferSizeExt(cusprHandle, P, dlPtr, dPtr, duPtr, realBuffer, M * N, P, &bufferSizeInBytes);
  if (tridSolverBuffer != nullptr) std::cerr << "The internal data have been initialized, be careful!\n";
  CHECK_CUDA_ERROR(hipMalloc(reinterpret_cast<void **>(&tridSolverBuffer), bufferSizeInBytes));
}

void gtsv2StridedBatch(hipsparseHandle_t handle, int m, const float *dl, const float *d, const float *du, float *x, int batchCount, int batchStride, void *pBuffer)
{
  CHECK_CUDA_ERROR(hipsparseSgtsv2StridedBatch(handle, m, dl, d, du, x, batchCount, batchStride, pBuffer));
}

void gtsv2StridedBatch(hipsparseHandle_t handle, int m, const double *dl, const double *d, const double *du, double *x, int batchCount, int batchStride, void *pBuffer)
{
  CHECK_CUDA_ERROR(hipsparseDgtsv2StridedBatch(handle, m, dl, d, du, x, batchCount, batchStride, pBuffer));
}

template <typename T>
void cufctSolver<T>::precondSolver(T *rhs)
{
  if (dlPtr == nullptr || dPtr == nullptr || duPtr == nullptr || tridSolverBuffer == nullptr) {
    std::cerr << "The internal data have not been initialized!\n";
    std::cerr << "There will be nothing to do in this routine.\n";
    return;
  }

  fctForward(rhs);

  int M{dims[0]}, N{dims[1]}, P{dims[2]};
  gtsv2StridedBatch(cusprHandle, P, dlPtr, dPtr, duPtr, rhs, M * N, P, tridSolverBuffer);

  fctBackward(rhs);
}

template <typename T>
void cuFreeMod(T *&ptr)
{
  if (ptr != nullptr) {
    CHECK_CUDA_ERROR(hipFree(ptr));
    ptr = nullptr;
  }
}

template <typename T>
cufctSolver<T>::~cufctSolver()
{
  cuFreeMod(tridSolverBuffer);
  cuFreeMod(duPtr);
  cuFreeMod(dPtr);
  cuFreeMod(dlPtr);

  CHECK_CUDA_ERROR(hipfftDestroy(c2rPlan));
  CHECK_CUDA_ERROR(hipfftDestroy(r2cPlan));

  cuFreeMod(compBuffer);
  cuFreeMod(realBuffer);
}

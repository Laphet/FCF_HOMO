#include "hip/hip_runtime.h"
#include "cuFctSolver.hpp"

template <typename T>
__global__ void fctPre(T *out, T const *in,
                       const int M, const int N, const int P)
{
  int idx{blockIdx.x * blockDim.x + threadIdx.x};
  int i{0}, j{0}, k{0}, buffer_head_idx{0}, idx_req{0};
  __shared__ T in_buffer[FCT_PRE_STENCIL_WIDTH]
                        [MAX_THREADS_PER_BLOCK +
                         PADDING_WIDTH_AVOID_BANK_CONFLICTS];

  if (idx < M * N * P)
  {
    get3dIdxFromIdx(i, j, k, idx, N, P);
    if (i < (M + 1) / 2 && j < (N + 1) / 2)
    {
      idx_req = getIdxFrom3dIdx(2 * i, 2 * j, k, N, P);
      buffer_head_idx = 0;
    }
    else if ((M + 1) / 2 <= i && j < (N + 1) / 2)
    {
      idx_req = getIdxFrom3dIdx(2 * M - 2 * i - 1, 2 * j, k, N, P);
      buffer_head_idx = 1;
    }
    else if (i < (M + 1) / 2 && (N + 1) / 2 <= j)
    {
      idx_req = getIdxFrom3dIdx(2 * i, 2 * N - 2 * j - 1, k, N, P);
      buffer_head_idx = 2;
    }
    else
    {
      idx_req = getIdxFrom3dIdx(2 * M - 2 * i - 1, 2 * N - 2 * j - 1,
                                k, N, P);
      buffer_head_idx = 3;
    }
    in_buffer[buffer_head_idx][threadIdx.x] = in[idx_req];
  }
  __syncthreads();

  if (idx < M * N * P)
  {
    out[idx] = in_buffer[buffer_head_idx][threadIdx.x];
  }
}

template <typename T>
__global__ void fctPost(T *out_hat, cuda::std::complex<T> const *in_hat,
                        const int M, const int N, const int P)
{
  using complex_t = cuda::std::complex<T>;
  int idx{blockIdx.x * blockDim.x + threadIdx.x};
  int i_p{0}, j_p{0}, k{0}, idx_req{0};
  __shared__ complex_t in_hat_buffer[FCT_POST_STENCIL_WIDTH]
                                    [MAX_THREADS_PER_BLOCK +
                                     PADDING_WIDTH_AVOID_BANK_CONFLICTS];

  if (idx < M * N * P)
  {
    get3dIdxFromIdx(i_p, j_p, k, idx, N, P);
    if (j_p <= N / 2)
    {
      idx_req = getIdxFrom3dIdxHalf(i_p, j_p, k, N, P);
      in_hat_buffer[0][threadIdx.x] = in_hat[idx_req];

      idx_req = getIdxFrom3dIdxHalf(M - i_p, j_p, k, N, P);
      in_hat_buffer[1][threadIdx.x] = in_hat[idx_req];
    }
    if (N / 2 + 1 <= j_p)
    {
      idx_req = getIdxFrom3dIdxHalf(M - i_p, N - j_p, k, N, P);
      in_hat_buffer[2][threadIdx.x] = in_hat[idx_req];

      idx_req = getIdxFrom3dIdxHalf(i_p, N - j_p, k, N, P);
      in_hat_buffer[3][threadIdx.x] = in_hat[idx_req];
    }
  }
  __syncthreads();

  T i_theta{static_cast<T>(0.0)}, j_theta{static_cast<T>(0.0)},
      cuPi{getPi<T>()};
  complex_t i_exp, j_exp, temp;

  if (idx < M * N * P)
  {
    i_theta = static_cast<T>(i_p) / static_cast(2 * M) * cuPi;
    j_theta = static_cast<T>(j_p) / static_cast(2 * N) * cuPi;
    i_exp = getExpItheta<T>(i_theta);
    j_exp = getExpItheta<T>(j_theta);
    if (1 <= i_p && 1 <= j_p && j_p <= N / 2)
    {
      temp = cuda::std::conj(j_exp) * in_hat_buffer[0][threadIdx.x];
      temp += j_exp * cuda::std::conj(in_hat_buffer[1][threadIdx.x]);
      temp *= cuda::std::conj(i_exp);
      out_hat[threadIdx.x] = temp.real() * static_cast<T>(0.5);
      return;
    }
    if (0 == i_p && 1 <= j_p && j_p <= N / 2)
    {
      temp = cuda::std::conj(j_exp) * in_hat_buffer[0][threadIdx.x];
      temp += j_exp * cuda::std::conj(in_hat_buffer[3][threadIdx.x]);
      out_hat[threadIdx.x] = temp.real() * static_cast<T>(0.5);
      return;
    }
    if (0 <= i_p && N / 2 + 1 <= j_p)
    {
      temp = cuda::std::conj(j_exp) *
             cuda::std::conj(in_hat_buffer[2][threadIdx.x]);
      temp += i_exp * in_hat_buffer[3][threadIdx.x];
      out_hat[threadIdx.x] = temp.real() * static_cast<T>(0.5);
      return;
    }
    if (0 == j_p)
    {
      temp = cuda::std::conj(i_exp) * in_hat_buffer[0][threadIdx.x];
      out_hat[threadIdx.x] = temp.real();
      return;
    }
  }
  else
    return;
}